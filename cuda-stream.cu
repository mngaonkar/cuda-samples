#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *data) {
    // Simple kernel that does nothing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = 0.13f;

}

int main() {
    float *d_data;
    float *h_data = (float*)malloc(1024 * sizeof(float));

    hipMalloc((void**)&d_data, 1024 * sizeof(float));
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    kernel<<<2, 512, 0, stream1>>>(d_data);
    hipStreamSynchronize(stream1);
    hipMemcpyAsync(h_data, d_data, 1024 * sizeof(float), hipMemcpyDeviceToHost, stream2);
    hipStreamSynchronize(stream2);

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
        hipFree(d_data);
        free(h_data);
        return -1;
    }

    // Verify results
    printf("Kernel executed successfully and data copied to host.\n");
    printf("Data[1022]: %f\n", h_data[1022]); // Print first
    printf("Data[1023]: %f\n", h_data[1023]); // Print second element to verify

    // Free resources
    hipFree(d_data);
    free(h_data);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
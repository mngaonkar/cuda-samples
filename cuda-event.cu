#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = 0.13f;
}

int main() {
    hipStream_t stream1, stream2;

    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *h_data = (float*)malloc(1024 * sizeof(float));
    float *d_data;
    hipMalloc((void**)&d_data, 1024 * sizeof(float));

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpyAsync(d_data, h_data, 1024 * sizeof(float), hipMemcpyHostToDevice, stream1);
    hipEventRecord(start, stream1);

    hipStreamWaitEvent(stream2, start);
    kernel<<<2, 512, 0, stream2>>>(d_data);
    hipEventRecord(stop, stream2);

    hipStreamWaitEvent(stream1, stop);
    hipMemcpyAsync(h_data, d_data, 1024 * sizeof(float), hipMemcpyDeviceToHost, stream1);

    hipStreamSynchronize(stream1);

    // Verify results
    printf("Kernel executed successfully and data copied to host.\n");
    printf("Data[1022]: %f\n", h_data[1022]); // Print first
    printf("Data[1023]: %f\n", h_data[1023]); // Print second element to verify

    // Free resources
    hipFree(d_data);
    free(h_data);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}